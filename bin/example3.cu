#include "event_group.hpp"

#include <cstdio>
#include <cstdlib>
#include <iostream>

int main(int argc, char **argv) {
  (void)argc;
  (void)argv;

  EventGroup group;

  // LLC reads/writes
  // group.add(PERF_TYPE_HW_CACHE, PERF_COUNT_HW_CACHE_LL |
  // (PERF_COUNT_HW_CACHE_OP_READ << 8) | (PERF_COUNT_HW_CACHE_RESULT_ACCESS <<
  // 16)); group.add(PERF_TYPE_HW_CACHE, PERF_COUNT_HW_CACHE_LL |
  // (PERF_COUNT_HW_CACHE_OP_WRITE << 8) | (PERF_COUNT_HW_CACHE_RESULT_ACCESS <<
  // 16));
    group.add(PERF_TYPE_HW_CACHE, PERF_COUNT_HW_CACHE_L1D |
                                      (PERF_COUNT_HW_CACHE_OP_WRITE << 8) |
                                      (PERF_COUNT_HW_CACHE_RESULT_ACCESS <<
                                      16));
  //   group.add(PERF_TYPE_HW_CACHE, PERF_COUNT_HW_CACHE_NODE |
  //                                     (PERF_COUNT_HW_CACHE_OP_READ << 8) |
  //                                     (PERF_COUNT_HW_CACHE_RESULT_ACCESS <<
  //                                     16));

  /*
  llc_references.pcie_read                          
       [PCIe read current. Derived from unc_c_tor_inserts.opcode. Unit: uncore_cbox]
        uncore_cbox_4/umask=0x1,event=0x35,filter_opc=0x19e/ 
  llc_references.pcie_write                         
       [PCIe write references (full cache line). Derived from unc_c_tor_inserts.opcode. Unit: uncore_cbox]
        uncore_cbox_4/umask=0x1,event=0x35,filter_opc=0x1c8,filter_tid=0x3e/ 
*/
//   group.add(PERF_TYPE_RAW, 0x35, 0x19e);
//   group.add(PERF_TYPE_RAW, 0xB0,0x08);
  group.add(PERF_TYPE_RAW, 0x51);
//   group.add(PERF_TYPE_RAW, 0x1c8, 0x35);

  hipFree(0);
  // TODO: cuda sets errno sometimes?
  errno = 0;

  char *dst, *src;
  hipMalloc(&dst, 1 << 20);
  hipHostAlloc(&src, 1 << 20, hipHostMallocDefault);

  //   group.reset();
  //   group.enable();

  for (int bytes = 1; bytes <= (1 << 20); bytes *= 2) {

    std::memset(src, bytes, 1);

    group.reset();
    group.enable();

    hipMemcpy(dst, src, bytes, hipMemcpyDefault);
    for (int i = 0; i < bytes; ++i) {
        src[i] = 0;
    }
    // std::memset(dst, 1, bytes);
    asm volatile("" ::"r,m"(src) : "memory");

    group.disable();

    std::cerr << bytes << ": ";

    std::vector<long long> vals = group.read();
    for (auto &e : vals) {
      std::cerr << e << " ";
    }
    std::cerr << "\n";
  }

  //   group.disable();

  //   std::vector<long long> vals = group.read();
  //   for (auto &e : vals) {
  //     std::cerr << e << "\n";
  //   }

//   hipFree(src);
//   hipHostFree(dst);
}